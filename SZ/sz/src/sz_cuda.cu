#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include "sz_cuda.h"
#include <conf.h>
#include <>
#include <math.h>
#include <printf.h>
#include <sqltypes.h>
#include <sz.h>
#include <sz_float.h>
#include <time.h>

// region CUDA macros
#ifdef __HIPCC__
#define GPU_DEVICE __device__
#define GPU_KERNEL __global__
#define GPU_HOST __host__
#define TIMER hipEvent_t
//TODO (robertu#1) when using full cuda instead of thrust, this version of parallel launch is required {{{
//#define PARALLEL_LAUNCH(num_blocks,num_threads,function_name) function_name<<<(num_blocks), (num_threads)>>>
#define PARALLEL_LAUNCH(num_blocks,num_threads,function_name) function_name
//END TODO (robertu#1) }}}
inline void
start_timer(TIMER* start)
{
  hipEventCreate(start);
  hipEventRecord(*start, 0);
}
/**
 * @param start  -- timer to stop and deallocate
 * @return the time elapsed in ms
 */
inline double
stop_timer(TIMER* start)
{
  hipEvent_t stop;
  hipEventCreate(&stop);
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float elapsed;
  hipEventElapsedTime(&elapsed, *start, stop);
  hipEventDestroy(stop);
  hipEventDestroy(*start);
  return elapsed;
}

#define NO_DEVICE -1

//TODO (robertu#2) use the openmp version until more cuda kernels are availible {{{
inline int
get_max_threads()
{
	return omp_get_max_threads();
}

inline void
set_max_threads(size_t threads)
{
  omp_set_num_threads(threads);
}

inline void
get_thread_id(int* idx)
{
	*idx = omp_get_thread_num();
}
/* code for cuda version when kernels are ready
inline int
get_max_threads()
{
  int num_devices;
  hipGetDeviceCount(&num_devices);
  for (int i = 0; i < num_devices; ++i) {
    struct hipDeviceProp_t properties;
    hipGetDeviceProperties(&properties, i);
    return properties.maxThreadsPerMultiProcessor;
  }
  return NO_DEVICE;
}
inline void
set_max_threads(size_t threads)
{
  (void)0; // NOOP on cuda silence compiler
}

void
get_thread_id(int* idx)
{
  *idx = blockIdx.x * blockDim.x + threadIdx.x;
}
*/
//END TODO (robertu#2) }}}

#else /*omp version*/
#define GPU_DEVICE
#define GPU_KERNEL
#define GPU_HOST
#define PARALLEL_LAUNCH(num_blocks,num_threads,function_name) function_name

#include <chrono>
#define TIMER std::chrono::high_resolution_clock::time_point

inline double stop_timer(TIMER* start)
{
	std::chrono::high_resolution_clock::time_point now = std::chrono::high_resolution_clock::now();
	std::chrono::duration<double, std::milli> duration = now - *start;
	return duration.count();
}

inline void start_timer(TIMER* start)
{
	*start = std::chrono::high_resolution_clock::now();
}

inline int
get_max_threads()
{
  return omp_get_max_threads();
}

inline void
set_max_threads(size_t threads)
{
  omp_set_num_threads(threads);
}

inline void
get_thread_id(int* idx)
{
  *idx = omp_get_thread_num();
}
#endif


// region Blockcount

struct BlockCount
{
  size_t split_index;
  size_t early_blockcount;
  size_t late_blockcount;
};
// endregion
// region MemoryBlocks
struct CompressionMemoryBlocks
{
  int* result_type;
  float* result_unpredictable_data;
  unsigned int* unpredictable_count;
  float* mean;
};

inline void
deleteCompressionMemoryBlocks(struct CompressionMemoryBlocks* memory)
{
  free((*memory).mean);
  free((*memory).result_unpredictable_data);
  free((*memory).unpredictable_count);
  free((*memory).result_type);
}

// endregion
// endregion
// region Prototypes


inline size_t get_offset(struct BlockCount const* const x, int const i);

inline struct CompressionMemoryBlocks
newCompressionMemoryBlocks(size_t num_blocks, size_t num_elements,
                           size_t unpred_data_max_size)
{
  struct CompressionMemoryBlocks memory;
  memory.result_type = (int*)malloc(num_elements * sizeof(int));
  memory.result_unpredictable_data =
    (float*)malloc(unpred_data_max_size * sizeof(float) * num_blocks);
  memory.unpredictable_count =
    (unsigned int*)malloc(num_blocks * sizeof(unsigned int));
  memory.mean = (float*)malloc(num_blocks * sizeof(float));
  return memory;
}

// endregion


unsigned char*
SZ_compress_float_1D_MDQ_cuda(float* oriData, size_t r1, double realPrecision,
                              size_t* comp_size)
{


  // determine quantization_intervals
  unsigned int quantization_intervals =
    quantization_intervals_1D(oriData, r1, realPrecision);

  // configure threads
  int thread_num = get_max_threads();
  size_t num_x = thread_num;
  set_max_threads(thread_num);

  // compute blockcounts
  struct BlockCount x;
  SZ_COMPUTE_BLOCKCOUNT(r1, num_x, x.split_index, x.early_blockcount,
                        x.late_blockcount);

  // compute work sizes
  size_t max_num_block_elements = x.early_blockcount;
  size_t num_blocks = num_x;
  size_t num_elements = r1;
  size_t unpred_data_max_size = max_num_block_elements;

  // allocate memory for compression
  struct CompressionMemoryBlocks memory =
    newCompressionMemoryBlocks(num_blocks, num_elements, unpred_data_max_size);

  // compress memory
  _sz_compress_float_1d_mdq_ra_block(oriData, r1, realPrecision, thread_num,
                                     unpred_data_max_size, &x, &memory);

  // build huffman encoding
  size_t nodeCount;
  unsigned char* treeBytes;
  unsigned int treeByteSize;
  buildHuffmanTree(thread_num, num_elements, &memory, &nodeCount, &treeBytes,
                   &treeByteSize);

  // compute number of unpredictable blocks
  size_t total_unpred = compute_total_unpred_gpu(num_blocks, &memory);

  // allocate output buffer
  unsigned char* result_pos;
  unsigned char* result = result_pos =
    (unsigned char*)malloc(compute_compressed_size(num_blocks, num_elements,
                                                   treeByteSize, total_unpred));

  // write compression metadata
  result_pos += initRandomAccessBytes(result_pos);
  result_pos = write_parallel_compresion_metadata(
    result_pos, thread_num, realPrecision, quantization_intervals, &memory,
    num_blocks, nodeCount, treeBytes, treeByteSize);
  free(treeBytes);

  // write unpredictable data
  size_t* unpred_offset =
    compute_unpred_offset(thread_num, num_blocks, &memory);
  copy_unpredictable(thread_num, unpred_data_max_size, &memory, result_pos,
                     unpred_offset);
  result_pos += total_unpred * sizeof(float);
  free(unpred_offset);

  // encode remaining data
  size_t* block_pos =
    (size_t*)result_pos; // block_pos exists to affect pointer math
  result_pos += num_blocks * sizeof(size_t);
  unsigned char* encoding_buffer =
    (unsigned char*)malloc(max_num_block_elements * sizeof(int) * num_blocks);
  encode_1D(oriData, thread_num, &x, max_num_block_elements, &memory, block_pos,
            encoding_buffer);

  // write encoded data
  size_t* block_offset =
    compute_block_offsets(thread_num, num_blocks, block_pos);
  copyEncodingBuffers(thread_num, max_num_block_elements, result_pos, block_pos,
                      encoding_buffer, block_offset);
  result_pos += block_offset[thread_num - 1] + block_pos[thread_num - 1];
  free(block_offset);

  // cleanup
  free(encoding_buffer);
  deleteCompressionMemoryBlocks(&memory);
  SZ_ReleaseHuffman();

  // return pointer to compressed data and size
  *comp_size = result_pos - result;
  return result;
}
unsigned char*
SZ_compress_float_2D_MDQ_cuda(float* oriData, size_t r1, size_t r2,
                              double realPrecision, size_t* comp_size)
{
  // determine quantization_intervals
  unsigned int quantization_intervals =
    quantization_intervals_2D(oriData, r1, r2, realPrecision);

  // configure threads
  int thread_num;
  size_t num_x;
  size_t num_y;
  config_threads_2D(&thread_num, &num_x, &num_y);

  // compute blockcounts
  struct BlockCount x, y;
  SZ_COMPUTE_BLOCKCOUNT(r1, num_x, x.split_index, x.early_blockcount,
                        x.late_blockcount);
  SZ_COMPUTE_BLOCKCOUNT(r2, num_y, y.split_index, y.early_blockcount,
                        y.late_blockcount);

  // compute work sizes
  size_t max_num_block_elements = x.early_blockcount * y.early_blockcount;
  size_t num_blocks = num_x * num_y;
  size_t num_elements = r1 * r2;
  size_t unpred_data_max_size = max_num_block_elements;
  size_t dim0_offset = r2;
  size_t buffer_size = y.early_blockcount * sizeof(float);

  // allocate memory for compression
  struct CompressionMemoryBlocks memory =
    newCompressionMemoryBlocks(num_blocks, num_elements, unpred_data_max_size);

  // compress memory
  PARALLEL_LAUNCH(1,thread_num,_sz_compress_float_2d_mdq_ra_block)(oriData, r1, r2, realPrecision, thread_num,
                                     num_y, unpred_data_max_size, dim0_offset,
                                     buffer_size, &x, &y, &memory);

  // build huffman encoding
  // TODO keep this one until I figure out why they pass arguments now
  // SZ_Reset(allNodes, stateNum);
  size_t nodeCount;
  unsigned char* treeBytes;
  unsigned int treeByteSize;
  buildHuffmanTree(thread_num, num_elements, &memory, &nodeCount, &treeBytes,
                   &treeByteSize);

  // compute number of unpredictable blocks
  size_t total_unpred = compute_total_unpred_gpu(num_blocks, &memory);

  // allocate output buffer
  unsigned char* result_pos;
  unsigned char* result = result_pos =
    (unsigned char*)malloc(compute_compressed_size(num_blocks, num_elements,
                                                   treeByteSize, total_unpred));

  // write compresion metadata
  result_pos += initRandomAccessBytes(result_pos);
  result_pos = write_parallel_compresion_metadata(
    result_pos, thread_num, realPrecision, quantization_intervals, &memory,
    num_blocks, nodeCount, treeBytes, treeByteSize);
  free(treeBytes);

  // write unpredictable data
  size_t* unpred_offset =
    compute_unpred_offset(thread_num, num_blocks, &memory);
  copy_unpredictable(thread_num, unpred_data_max_size, &memory, result_pos,
                     unpred_offset);
  result_pos += total_unpred * sizeof(float);
  free(unpred_offset);

  // encode remaining data
  size_t* block_pos = (size_t*)result_pos;
  result_pos += num_blocks * sizeof(size_t);
  unsigned char* encoding_buffer =
    (unsigned char*)malloc(max_num_block_elements * sizeof(int) * num_blocks);
  PARALLEL_LAUNCH(1,thread_num,encode_2D)(thread_num, num_y, &x, &y, max_num_block_elements, dim0_offset,
            &memory, block_pos, encoding_buffer);

  // write encoded data
  size_t* block_offset =
    compute_block_offsets(thread_num, num_blocks, block_pos);
  copyEncodingBuffers(thread_num, max_num_block_elements, result_pos, block_pos,
                      encoding_buffer, block_offset);
  result_pos += block_offset[thread_num - 1] + block_pos[thread_num - 1];
  free(block_offset);

  // cleanup
  free(encoding_buffer);
  deleteCompressionMemoryBlocks(&memory);
  SZ_ReleaseHuffman();

  *comp_size = result_pos - result;
  return result;
}
unsigned char*
SZ_compress_float_3D_MDQ_cuda(float* oriData, size_t r1, size_t r2, size_t r3,
                              double realPrecision, size_t* comp_size)
{


  // determine quantization_intervals
  unsigned int quantization_intervals =
    quantization_intervals_3D(oriData, r1, r2, r3, realPrecision);

  // configure threads
  int thread_num;
  size_t num_x, num_y, num_z;
  config_threads_3D(&thread_num, &num_x, &num_y, &num_z);

  // compute blockcounts
  struct BlockCount x, y, z;
  SZ_COMPUTE_BLOCKCOUNT(r1, num_x, x.split_index, x.early_blockcount,
                        x.late_blockcount);
  SZ_COMPUTE_BLOCKCOUNT(r2, num_y, y.split_index, y.early_blockcount,
                        y.late_blockcount);
  SZ_COMPUTE_BLOCKCOUNT(r3, num_z, z.split_index, z.early_blockcount,
                        z.late_blockcount);

  // compute work sizes
  size_t max_num_block_elements =
    x.early_blockcount * y.early_blockcount * z.early_blockcount;
  size_t num_blocks = num_x * num_y * num_z;
  size_t num_elements = r1 * r2 * r3;
  size_t unpred_data_max_size = max_num_block_elements;
  size_t dim0_offset = r2 * r3;
  size_t dim1_offset = r3;
  int num_yz = num_y * num_z;
  size_t buffer_size = y.early_blockcount * z.early_blockcount * sizeof(float);

  // allocate memory for compression
  struct CompressionMemoryBlocks memory =
    newCompressionMemoryBlocks(num_blocks, num_elements, unpred_data_max_size);

  // compress memory
  PARALLEL_LAUNCH(1,thread_num,_sz_compress_float_3d_mdq_ra_block)(
    oriData, r1, r2, r3, realPrecision, thread_num, num_z, unpred_data_max_size,
    dim0_offset, dim1_offset, num_yz, buffer_size, &x, &y, &z, &memory);

  // build huffman encoding
  size_t nodeCount;
  unsigned char* treeBytes;
  unsigned int treeByteSize;
  buildHuffmanTree(thread_num, num_elements, &memory, &nodeCount, &treeBytes,
                   &treeByteSize);

  // compute number of unpredictable blocks
  size_t total_unpred = compute_total_unpred_gpu(num_blocks, &memory);

  // allocate output buffer
  unsigned char* result_pos;
  unsigned char* result = result_pos =
    (unsigned char*)malloc(compute_compressed_size(num_blocks, num_elements,
                                                   treeByteSize, total_unpred));

  // write compression metadata
  result_pos += initRandomAccessBytes(result_pos);
  result_pos = write_parallel_compresion_metadata(
    result_pos, thread_num, realPrecision, quantization_intervals, &memory,
    num_blocks, nodeCount, treeBytes, treeByteSize);
  free(treeBytes);

  // write unpredictable data
  size_t* unpred_offset =
    compute_unpred_offset(thread_num, num_blocks, &memory);
  copy_unpredictable(thread_num, unpred_data_max_size, &memory, result_pos,
                     unpred_offset);
  result_pos += total_unpred * sizeof(float);
  free(unpred_offset);

  // encode remaining data
  size_t* block_pos = (size_t*)result_pos;
  result_pos += num_blocks * sizeof(size_t);
  unsigned char* encoding_buffer =
    (unsigned char*)malloc(max_num_block_elements * sizeof(int) * num_blocks);
  PARALLEL_LAUNCH(1,thread_num,encode_3D)(thread_num, num_z, &x, &y, &z, max_num_block_elements, dim0_offset,
            dim1_offset, num_yz, &memory, block_pos, encoding_buffer);

  // write encoded data
  size_t* block_offset =
    compute_block_offsets(thread_num, num_blocks, block_pos);
  copyEncodingBuffers(thread_num, max_num_block_elements, result_pos, block_pos,
                      encoding_buffer, block_offset);
  result_pos += block_offset[thread_num - 1] + block_pos[thread_num - 1];
  free(block_offset);

  // cleanup
  free(encoding_buffer);
  deleteCompressionMemoryBlocks(&memory);
  SZ_ReleaseHuffman();

  *comp_size = result_pos - result;
  return result;
}


void
decompressDataSeries_float_1D_cuda(float** data, size_t r1,
                                   unsigned char* comp_data)
{

  size_t num_elements = r1;

  *data = (float*)malloc(sizeof(float) * num_elements);

  unsigned char* comp_data_pos = comp_data;
  int thread_num = readIntBigEndian(&comp_data_pos);
  size_t num_x = thread_num;

  set_max_threads(thread_num);
  struct BlockCount x;
  SZ_COMPUTE_BLOCKCOUNT(r1, num_x, x.split_index, x.early_blockcount,
                        x.late_blockcount);

  size_t num_blocks = num_x;

  double realPrecision = bytesToDouble(comp_data_pos);
  comp_data_pos += 8;
  unsigned int intervals = readIntBigEndian(&comp_data_pos);

  updateQuantizationInfo(intervals);
  // intvRadius = (int)((tdps->intervals - 1)/ 2);

  struct CompressionMemoryBlocks memory;
  unsigned int tree_size = readIntBigEndian(&comp_data_pos);
  allNodes = readIntBigEndian(&comp_data_pos);
  stateNum = allNodes / 2;
  SZ_Reset();
  node root =
    reconstruct_HuffTree_from_bytes_anyStates(comp_data_pos, allNodes);
  comp_data_pos += tree_size;

  unsigned int* unpred_count = (unsigned int*)comp_data_pos;
  comp_data_pos += num_blocks * sizeof(unsigned int);

  float* mean_pos = (float*)comp_data_pos;
  comp_data_pos += num_blocks * sizeof(float);

  memory.result_unpredictable_data = (float*)comp_data_pos;

  size_t total_unpred = 0;
  size_t* unpred_offset = (size_t*)malloc(num_blocks * sizeof(size_t));
  for (int i = 0; i < num_blocks; i++) {
    unpred_offset[i] = total_unpred;
    total_unpred += unpred_count[i];
  }

  comp_data_pos += total_unpred * sizeof(float);

  memory.result_type = (int*)malloc(num_elements * sizeof(int));
  // decode(comp_data_pos, num_elements, root, memory.result_type);
  size_t* block_offset = (size_t*)malloc(num_blocks * sizeof(size_t));
  size_t* block_pos = (size_t*)comp_data_pos;
  comp_data_pos += num_blocks * sizeof(size_t);
  block_offset[0] = 0;
  for (int t = 1; t < thread_num; t++) {
    block_offset[t] = block_pos[t - 1] + block_offset[t - 1];
  }
#pragma omp parallel for
  for (int t = 0; t < thread_num; t++) {
    int i = t;
    size_t offset_x = get_offset(&x, i);
    size_t current_blockcount_x = get_current_blockcount(&x, i);
    size_t type_offset = offset_x;
    int* type = memory.result_type + type_offset;
    decode(comp_data_pos + block_offset[t], current_blockcount_x, root, type);
  }

#pragma omp parallel for
  for (int t = 0; t < thread_num; t++) {
    int i = t;
    size_t offset_x = get_offset(&x, i);
    float* data_pos = *data + offset_x;

    size_t current_blockcount_x = get_current_blockcount(&x, i);
    size_t type_offset = offset_x;
    int* type = memory.result_type + type_offset;

    float* unpredictable_data =
      memory.result_unpredictable_data + unpred_offset[t];
    float mean = mean_pos[t];
    int cur_unpred_count = decompressDataSeries_float_1D_RA_block(
      data_pos, mean, r1, current_blockcount_x, realPrecision, type,
      unpredictable_data);
  }

  free(memory.result_type);
  free(unpred_offset);
}

void
decompressDataSeries_float_2D_cuda(float** data, size_t r1, size_t r2,
                                   unsigned char* comp_data)
{
  // printf("num_block_elements %d num_blocks %d\n", max_num_block_elements,
  // num_blocks); fflush(stdout);
  TIMER timer;
  start_timer(&timer);

  size_t dim0_offset = r2;
  size_t num_elements = r1 * r2;

  *data = (float*)malloc(sizeof(float) * num_elements);

  unsigned char* comp_data_pos = comp_data;

  int thread_num = bytesToInt_bigEndian(comp_data_pos);
  comp_data_pos += 4;
  int thread_order = (int)log2(thread_num);
  size_t num_x, num_y;
  {
    int block_thread_order = thread_order / 2;
    switch (thread_order % 2) {
      case 0: {
        num_x = 1 << block_thread_order;
        num_y = 1 << block_thread_order;
        break;
      }
      case 1: {
        num_x = 1 << (block_thread_order + 1);
        num_y = 1 << block_thread_order;
        break;
      }
    }
  }
  printf("number of blocks: %zu %zu, thread_num %d\n", num_x, num_y,
         thread_num);
  set_max_threads(thread_num);
  struct BlockCount x, y;
  SZ_COMPUTE_BLOCKCOUNT(r1, num_x, x.split_index, x.early_blockcount,
                        x.late_blockcount);
  SZ_COMPUTE_BLOCKCOUNT(r2, num_y, y.split_index, y.early_blockcount,
                        y.late_blockcount);

  size_t num_blocks = num_x * num_y;

  double realPrecision = bytesToDouble(comp_data_pos);
  comp_data_pos += 8;
  unsigned int intervals = bytesToInt_bigEndian(comp_data_pos);
  comp_data_pos += 4;

  updateQuantizationInfo(intervals);
  // intvRadius = (int)((tdps->intervals - 1)/ 2);

  unsigned int tree_size = bytesToInt_bigEndian(comp_data_pos);
  comp_data_pos += 4;
  allNodes = bytesToInt_bigEndian(comp_data_pos);
  stateNum = allNodes / 2;
  SZ_Reset();
  // printf("Reconstruct huffman tree with node count %ld\n", nodeCount);
  // fflush(stdout);
  node root =
    reconstruct_HuffTree_from_bytes_anyStates(comp_data_pos + 4, allNodes);

  struct CompressionMemoryBlocks memory;
  comp_data_pos += 4 + tree_size;
  unsigned int* unpred_count = (unsigned int*)comp_data_pos;
  comp_data_pos += num_blocks * sizeof(unsigned int);
  float* mean_pos = (float*)comp_data_pos;
  comp_data_pos += num_blocks * sizeof(float);
  memory.result_unpredictable_data = (float*)comp_data_pos;
  size_t total_unpred = 0;
  size_t* unpred_offset = (size_t*)malloc(num_blocks * sizeof(size_t));
  for (int i = 0; i < num_blocks; i++) {
    unpred_offset[i] = total_unpred;
    total_unpred += unpred_count[i];
  }
  comp_data_pos += total_unpred * sizeof(float);

  memory.result_type = (int*)malloc(num_elements * sizeof(int));
  // decode(comp_data_pos, num_elements, root, memory.result_type);
  size_t* block_offset = (size_t*)malloc(num_blocks * sizeof(size_t));
  size_t* block_pos = (size_t*)comp_data_pos;
  comp_data_pos += num_blocks * sizeof(size_t);
  block_offset[0] = 0;
  for (int t = 1; t < thread_num; t++) {
    block_offset[t] = block_pos[t - 1] + block_offset[t - 1];
  }
  printf("Read data info elapsed time: %.4f\n", stop_timer(&timer));
  start_timer(&timer);
#pragma omp parallel for
  for (int t = 0; t < thread_num; t++) {
    int i = t / (num_y);
    int j = (t % num_y);
    size_t offset_x = get_offset(&x, i);
    size_t offset_y = get_offset(&y, j);

    size_t current_blockcount_x = get_current_blockcount(&x, i);
    size_t current_blockcount_y = get_current_blockcount(&y, j);

    size_t type_offset =
      offset_x * dim0_offset + offset_y * current_blockcount_x;
    int* type = memory.result_type + type_offset;
    decode(comp_data_pos + block_offset[t],
           current_blockcount_x * current_blockcount_y, root, type);
  }
  printf("Parallel Huffman decoding elapsed time: %.4f\n", stop_timer(&timer));
  start_timer(&timer);

#pragma omp parallel for
  for (int t = 0; t < thread_num; t++) {
    int i = t / num_y;
    int j = (t % num_y);
    // printf("%d: %d %d %d\n", omp_get_thread_num(), i, j, k);
    size_t offset_x = get_offset(&x, i);
    size_t offset_y = get_offset(&y, j);
    float* data_pos = *data + offset_x * dim0_offset + offset_y;

    size_t current_blockcount_x = get_current_blockcount(&x, i);
    size_t current_blockcount_y = get_current_blockcount(&y, j);

    size_t type_offset =
      offset_x * dim0_offset + offset_y * current_blockcount_x;
    int* type = memory.result_type + type_offset;

    float* unpredictable_data =
      memory.result_unpredictable_data + unpred_offset[t];
    float mean = mean_pos[t];
    // printf("\n%d\ndata_offset: %ld\n", t, offset_x * dim0_offset + offset_y *
    // dim1_offset + offset_z); printf("memory.mean: %.2f\n", memory.mean);
    // for(int tmp=0; tmp<10; tmp++){
    // 	printf("%.2f ", unpredictable_data[tmp]);
    // }
    // printf("\n\n");
    int cur_unpred_count = decompressDataSeries_float_2D_RA_block(
      data_pos, mean, r1, r2, current_blockcount_x, current_blockcount_y,
      realPrecision, type, unpredictable_data);
  }
  printf("Parallel decompress elapsed time: %.4f\n", stop_timer(&timer));

  free(memory.result_type);
  free(unpred_offset);
}
void
decompressDataSeries_float_3D_cuda(float** data, size_t r1, size_t r2,
                                   size_t r3, unsigned char* comp_data)
{
  // printf("num_block_elements %d num_blocks %d\n", max_num_block_elements,
  // num_blocks); fflush(stdout);
  double elapsed_time = 0.0;
  elapsed_time = -omp_get_wtime();

  size_t dim0_offset = r2 * r3;
  size_t dim1_offset = r3;
  size_t num_elements = r1 * r2 * r3;

  *data = (float*)malloc(sizeof(float) * num_elements);

  unsigned char* comp_data_pos = comp_data;
  // int meta_data_offset = 3 + 1 + MetaDataByteLength;
  // comp_data_pos += meta_data_offset;

  int thread_num = bytesToInt_bigEndian(comp_data_pos);
  comp_data_pos += 4;
  int thread_order = (int)log2(thread_num);
  size_t num_x, num_y, num_z;
  {
    int block_thread_order = thread_order / 3;
    switch (thread_order % 3) {
      case 0: {
        num_x = 1 << block_thread_order;
        num_y = 1 << block_thread_order;
        num_z = 1 << block_thread_order;
        break;
      }
      case 1: {
        num_x = 1 << (block_thread_order + 1);
        num_y = 1 << block_thread_order;
        num_z = 1 << block_thread_order;
        break;
      }
      case 2: {
        num_x = 1 << (block_thread_order + 1);
        num_y = 1 << (block_thread_order + 1);
        num_z = 1 << block_thread_order;
        break;
      }
    }
  }
  printf("number of blocks: %zu %zu %zu, thread_num %d\n", num_x, num_y, num_z,
         thread_num);
  set_max_threads(thread_num);
  struct BlockCount x, y, z;
  SZ_COMPUTE_BLOCKCOUNT(r1, num_x, x.split_index, x.early_blockcount,
                        x.late_blockcount);
  SZ_COMPUTE_BLOCKCOUNT(r2, num_y, y.split_index, y.early_blockcount,
                        y.late_blockcount);
  SZ_COMPUTE_BLOCKCOUNT(r3, num_z, z.split_index, z.early_blockcount,
                        z.late_blockcount);

  size_t num_blocks = num_x * num_y * num_z;

  double realPrecision = bytesToDouble(comp_data_pos);
  comp_data_pos += 8;
  unsigned int intervals = bytesToInt_bigEndian(comp_data_pos);
  comp_data_pos += 4;

  updateQuantizationInfo(intervals);
  // intvRadius = (int)((tdps->intervals - 1)/ 2);

  unsigned int tree_size = bytesToInt_bigEndian(comp_data_pos);
  comp_data_pos += 4;
  allNodes = bytesToInt_bigEndian(comp_data_pos);
  stateNum = allNodes / 2;
  SZ_Reset();
  // printf("Reconstruct huffman tree with node count %ld\n", nodeCount);
  // fflush(stdout);
  node root =
    reconstruct_HuffTree_from_bytes_anyStates(comp_data_pos + 4, allNodes);

  struct CompressionMemoryBlocks memory;
  comp_data_pos += 4 + tree_size;
  unsigned int* unpred_count = (unsigned int*)comp_data_pos;
  comp_data_pos += num_blocks * sizeof(unsigned int);
  float* mean_pos = (float*)comp_data_pos;
  comp_data_pos += num_blocks * sizeof(float);
  memory.result_unpredictable_data = (float*)comp_data_pos;
  size_t total_unpred = 0;
  size_t* unpred_offset = (size_t*)malloc(num_blocks * sizeof(size_t));
  for (int i = 0; i < num_blocks; i++) {
    unpred_offset[i] = total_unpred;
    total_unpred += unpred_count[i];
  }
  comp_data_pos += total_unpred * sizeof(float);

  memory.result_type = (int*)malloc(num_elements * sizeof(int));
  // decode(comp_data_pos, num_elements, root, memory.result_type);
  size_t* block_offset = (size_t*)malloc(num_blocks * sizeof(size_t));
  size_t* block_pos = (size_t*)comp_data_pos;
  comp_data_pos += num_blocks * sizeof(size_t);
  block_offset[0] = 0;
  for (int t = 1; t < thread_num; t++) {
    block_offset[t] = block_pos[t - 1] + block_offset[t - 1];
  }
  int num_yz = num_y * num_z;
  elapsed_time += omp_get_wtime();
  printf("Read data info elapsed time: %.4f\n", elapsed_time);
  elapsed_time = -omp_get_wtime();

#pragma omp parallel for
  for (int t = 0; t < thread_num; t++) {
    int i = t / (num_yz);
    int j = (t % num_yz) / num_z;
    int k = t % num_z;
    size_t offset_x = get_offset(&x, i);
    size_t offset_y = get_offset(&y, j);
    size_t offset_z = get_offset(&z, k);
    size_t current_blockcount_x = get_current_blockcount(&x, i);
    size_t current_blockcount_y = get_current_blockcount(&y, j);
    size_t current_blockcount_z = get_current_blockcount(&z, k);
    size_t type_offset = offset_x * dim0_offset +
                         offset_y * current_blockcount_x * dim1_offset +
                         offset_z * current_blockcount_x * current_blockcount_y;
    int* type = memory.result_type + type_offset;
    decode(comp_data_pos + block_offset[t],
           current_blockcount_x * current_blockcount_y * current_blockcount_z,
           root, type);
  }
  elapsed_time += omp_get_wtime();
  printf("Parallel Huffman decoding elapsed time: %.4f\n", elapsed_time);
  elapsed_time = -omp_get_wtime();

#pragma omp parallel for
  for (int t = 0; t < thread_num; t++) {
    int i = t / (num_yz);
    int j = (t % num_yz) / num_z;
    int k = t % num_z;
    // printf("%d: %d %d %d\n", omp_get_thread_num(), i, j, k);
    size_t offset_x = get_offset(&x, i);
    size_t offset_y = get_offset(&y, j);
    size_t offset_z = get_offset(&z, k);

    float* data_pos =
      *data + offset_x * dim0_offset + offset_y * dim1_offset + offset_z;

    size_t current_blockcount_x = get_current_blockcount(&x, i);
    size_t current_blockcount_y = get_current_blockcount(&y, j);
    size_t current_blockcount_z = get_current_blockcount(&z, k);

    size_t type_offset = offset_x * dim0_offset +
                         offset_y * current_blockcount_x * dim1_offset +
                         offset_z * current_blockcount_x * current_blockcount_y;
    int* type = memory.result_type + type_offset;

    float* unpredictable_data =
      memory.result_unpredictable_data + unpred_offset[t];
    float mean = mean_pos[t];
    int cur_unpred_count = decompressDataSeries_float_3D_RA_block(
      data_pos, mean, r1, r2, r3, current_blockcount_x, current_blockcount_y,
      current_blockcount_z, realPrecision, type, unpredictable_data);
  }
  elapsed_time += omp_get_wtime();
  printf("Parallel decompress elapsed time: %.4f\n", elapsed_time);

  free(memory.result_type);
  free(unpred_offset);
}



inline void
_sz_compress_float_1d_mdq_ra_block(float* oriData, size_t r1,
                                   double realPrecision, int thread_num,
                                   size_t unpred_data_max_size,
                                   struct BlockCount* x,
                                   struct CompressionMemoryBlocks* memory)
{
  #pragma omp parallel for
  for (int t = 0; t < thread_num; t++) {

	  int id;
	  get_thread_id(&id);
    size_t offset_x = get_offset(x, id);
    size_t current_blockcount_x = get_current_blockcount(x, id);

    float* data_pos = oriData + offset_x;

    size_t type_offset = offset_x;
    int* type = (*memory).result_type + type_offset;

    float* unpredictable_data =
      (*memory).result_unpredictable_data + id * unpred_data_max_size;
    (*memory).unpredictable_count[id] = SZ_compress_float_1D_MDQ_RA_block(
      data_pos, (*memory).mean + id, r1, current_blockcount_x, realPrecision,
      type, unpredictable_data);
  }
}

inline void
_sz_compress_float_2d_mdq_ra_block(float* oriData, size_t r1, size_t r2,
                                   double realPrecision, int thread_num,
                                   size_t num_y, size_t unpred_data_max_size,
                                   size_t dim0_offset, size_t buffer_size,
                                   struct BlockCount* x, struct BlockCount* y,
                                   struct CompressionMemoryBlocks* memory)
{
  float *P0, *P1; // buffer
  P0 = (float*)malloc(buffer_size * thread_num);
  P1 = (float*)malloc(buffer_size * thread_num);
#pragma omp parallel for
  for (int t = 0; t < thread_num; t++) {
    int i = t / (num_y);
    int j = (t % num_y);
    size_t offset_x = get_offset(x, i);
    size_t offset_y = get_offset(x, j);
    float* data_pos = oriData + offset_x * dim0_offset + offset_y;

    size_t current_blockcount_x = get_current_blockcount(x, i);
    size_t current_blockcount_y = get_current_blockcount(y, j);
    size_t type_offset =
      offset_x * dim0_offset + offset_y * current_blockcount_x;
    int* type = (*memory).result_type + type_offset;

    float* unpredictable_data =
      (*memory).result_unpredictable_data + t * unpred_data_max_size;
    (*memory).unpredictable_count[t] = SZ_compress_float_2D_MDQ_RA_block(
      data_pos, (*memory).mean + t, r1, r2, current_blockcount_x,
      current_blockcount_y, realPrecision, P0 + (t * buffer_size),
      P1 + (t * buffer_size), type, unpredictable_data);
  }
  free(P0);
  free(P1);
}

inline void
_sz_compress_float_3d_mdq_ra_block(
  float* oriData, size_t r1, size_t r2, size_t r3, double realPrecision,
  int thread_num, size_t num_z, size_t unpred_data_max_size, size_t dim0_offset,
  size_t dim1_offset, int num_yz, size_t buffer_size, struct BlockCount* x,
  struct BlockCount* y, struct BlockCount* z,
  struct CompressionMemoryBlocks* memory)
{
  float* P0 = (float*)malloc(buffer_size * thread_num);
  float* P1 = (float*)malloc(buffer_size * thread_num);
#pragma omp parallel for
  for (int t = 0; t < thread_num; t++) {
    int i = t / (num_yz);
    int j = (t % num_yz) / num_z;
    int k = t % num_z;
    size_t offset_x = get_offset(x, i);
    size_t offset_y = get_offset(y, j);
    size_t offset_z = get_offset(z, k);

    float* data_pos =
      oriData + offset_x * dim0_offset + offset_y * dim1_offset + offset_z;

    size_t current_blockcount_x = get_current_blockcount(x, i);
    size_t current_blockcount_y = get_current_blockcount(y, j);
    size_t current_blockcount_z = get_current_blockcount(z, k);

    size_t type_offset = offset_x * dim0_offset +
                         offset_y * current_blockcount_x * dim1_offset +
                         offset_z * current_blockcount_x * current_blockcount_y;

    int* type = (*memory).result_type + type_offset;

    float* unpredictable_data =
      (*memory).result_unpredictable_data + t * unpred_data_max_size;
    (*memory).unpredictable_count[t] = SZ_compress_float_3D_MDQ_RA_block(
      data_pos, (*memory).mean + t, r1, r2, r3, current_blockcount_x,
      current_blockcount_y, current_blockcount_z, realPrecision,
      P0 + (t * buffer_size), P1 + (t * buffer_size), type,
      unpredictable_data);
  }
  free(P0);
  free(P1);
}
void
buildHuffmanTree(int thread_num, size_t num_elements,
                 struct CompressionMemoryBlocks* memory, size_t* nodeCount,
                 unsigned char** treeBytes, unsigned int* treeByteSize)
{
  (*nodeCount) = 0;
  SZ_Reset();
  Huffman_init_cuda((*memory).result_type, num_elements, thread_num);
  for (size_t i = 0; i < stateNum; i++)
    if (code[i])
      (*nodeCount)++;
  (*nodeCount) = (*nodeCount) * 2 - 1;
  *treeByteSize =
    convert_HuffTree_to_bytes_anyStates((int)*nodeCount, treeBytes);
}

inline void
copy_unpredictable(int thread_num, size_t unpred_data_max_size,
                   struct CompressionMemoryBlocks* memory,
                   unsigned char* result_pos, const size_t* unpred_offset)
{
#pragma omp parallel for
  for (int t = 0; t < thread_num; t++) {
    float* unpredictable_data =
      (*memory).result_unpredictable_data + t * unpred_data_max_size;
    memcpy(result_pos + unpred_offset[t] * sizeof(float), unpredictable_data,
           (*memory).unpredictable_count[t] * sizeof(float));
  }
}

inline void
copyEncodingBuffers(int thread_num, size_t max_num_block_elements,
                    unsigned char* result_pos, size_t* block_pos,
                    unsigned char* encoding_buffer, const size_t* block_offset)
{
  #pragma omp parallel for
  for (int t = 0; t < thread_num; t++) {
    memcpy(result_pos + block_offset[t],
           encoding_buffer + t * max_num_block_elements * sizeof(int),
           block_pos[t]);
  }
}
inline size_t
compute_compressed_size(size_t num_blocks, size_t num_elements,
                        unsigned int treeByteSize, size_t total_unpred)
{

  unsigned int meta_data_offset = 3 + 1 + MetaDataByteLength;
  return meta_data_offset +                    // metadata
         sizeof(double) +                      // real precision
         sizeof(int) +                         // intervals
         sizeof(int) +                         // nodeCount
         treeByteSize +                        // huffman
         num_blocks * sizeof(unsigned short) + // block index
         num_blocks * sizeof(unsigned short) + // unpredictable count
         num_blocks * sizeof(float) +          // memory.mean
         total_unpred * sizeof(float) +        // unpred
         num_elements * sizeof(int);           // elements
}

inline size_t
compute_total_unpred(size_t num_blocks, struct CompressionMemoryBlocks* memory)
{
  size_t total_unpred = 0;
#pragma omp parallel for reduction(+:total_unpred)
  for (int i = 0; i < num_blocks; i++) {
    total_unpred += (*memory).unpredictable_count[i];
  }
  return total_unpred;
}

inline size_t
compute_total_unpred_gpu(size_t num_blocks, struct CompressionMemoryBlocks* memory)
{
	thrust::device_vector<unsigned int> dev(memory->unpredictable_count, memory->unpredictable_count + num_blocks);
	return thrust::reduce(dev.begin(), dev.end());
}

inline size_t*
compute_unpred_offset(int thread_num, size_t num_blocks,
                      struct CompressionMemoryBlocks* memory)
{
  size_t* unpred_offset = (size_t*)malloc(num_blocks * sizeof(size_t));
  unpred_offset[0] = 0;
  for (int t = 1; t < thread_num; t++) {
    unpred_offset[t] =
      (*memory).unpredictable_count[t - 1] + unpred_offset[t - 1];
  }
  return unpred_offset;
}

inline size_t*
compute_block_offsets(int thread_num, size_t num_blocks,
                      const size_t* block_pos)
{
  size_t* block_offset = (size_t*)malloc(num_blocks * sizeof(size_t));
  block_offset[0] = 0;
  for (int t = 1; t < thread_num; t++) {
    block_offset[t] = block_pos[t - 1] + block_offset[t - 1];
  }
  return block_offset;
}

inline void
config_threads_2D(int* thread_num, size_t* num_x, size_t* num_y)
{
  (*thread_num) = omp_get_max_threads();
  int thread_order = (int)log2((*thread_num));
  {
    int block_thread_order = thread_order / 2;
    switch (thread_order % 2) {
      case 0: {
        (*num_x) = 1 << block_thread_order;
        (*num_y) = 1 << block_thread_order;
        break;
      }
      case 1: {
        (*num_x) = 1 << (block_thread_order + 1);
        (*num_y) = 1 << block_thread_order;
        break;
      }
    }
    (*thread_num) = (*num_x) * (*num_y);
  }
  set_max_threads((*thread_num));
  // calculate block dims
}

inline void
config_threads_3D(int* thread_num, size_t* num_x, size_t* num_y, size_t* num_z)
{
  (*thread_num) = omp_get_max_threads();
  int thread_order = (int)log2((*thread_num));
  {
    int block_thread_order = thread_order / 3;
    switch (thread_order % 3) {
      case 0: {
        (*num_x) = 1 << block_thread_order;
        (*num_y) = 1 << block_thread_order;
        (*num_z) = 1 << block_thread_order;
        break;
      }
      case 1: {
        (*num_x) = 1 << (block_thread_order + 1);
        (*num_y) = 1 << block_thread_order;
        (*num_z) = 1 << block_thread_order;
        break;
      }
      case 2: {
        (*num_x) = 1 << (block_thread_order + 1);
        (*num_y) = 1 << (block_thread_order + 1);
        (*num_z) = 1 << block_thread_order;
        break;
      }
    }
    (*thread_num) = (*num_x) * (*num_y) * (*num_z);
  }
  set_max_threads((*thread_num));
}

inline size_t
get_current_blockcount(struct BlockCount const* const x, int const i)
{
  return ((i < x->split_index) ? x->early_blockcount : x->late_blockcount);
}
inline size_t
get_offset(struct BlockCount const* const x, int const i)
{
  return ((i < x->split_index) ? i * x->early_blockcount
                               : i * x->late_blockcount + x->split_index);
}

inline void
encode_1D(float* oriData, int thread_num, struct BlockCount* x,
          size_t max_num_block_elements, struct CompressionMemoryBlocks* memory,
          size_t* block_pos, unsigned char* encoding_buffer)
{
#pragma omp parallel for
  for (int t = 0; t < thread_num; t++) {
    int i = t;
    unsigned char* encoding_buffer_pos =
      encoding_buffer + t * max_num_block_elements * sizeof(int);
    size_t enCodeSize = 0;
    size_t offset_x = get_offset(x, i);
    size_t current_blockcount_x = get_current_blockcount(x, i);
    size_t current_block_elements = current_blockcount_x;
    size_t type_offset = offset_x;
    int* type = (*memory).result_type + type_offset;
    encode(type, current_block_elements, encoding_buffer_pos, &enCodeSize);
    block_pos[t] = enCodeSize;
  }
}
inline void
encode_2D(int thread_num, size_t num_y, struct BlockCount* x,
          struct BlockCount* y, size_t max_num_block_elements,
          size_t dim0_offset, struct CompressionMemoryBlocks* memory,
          size_t* block_pos, unsigned char* encoding_buffer)
{
#pragma omp parallel for
  for (int t = 0; t < thread_num; t++) {
    int i = t / (num_y);
    int j = (t % num_y);
    unsigned char* encoding_buffer_pos =
      encoding_buffer + t * max_num_block_elements * sizeof(int);
    size_t enCodeSize = 0;

    size_t offset_x = get_offset(x, i);
    size_t offset_y = get_offset(y, j);

    size_t current_blockcount_x = get_current_blockcount(x, i);
    size_t current_blockcount_y = get_current_blockcount(y, j);

    size_t current_block_elements = current_blockcount_x * current_blockcount_y;
    size_t type_offset =
      offset_x * dim0_offset + offset_y * current_blockcount_x;
    int* type = (*memory).result_type + type_offset;
    encode(type, current_block_elements, encoding_buffer_pos, &enCodeSize);
    block_pos[t] = enCodeSize;
  }
}
inline void
encode_3D(int thread_num, size_t num_z, struct BlockCount* x,
          struct BlockCount* y, struct BlockCount* z,
          size_t max_num_block_elements, size_t dim0_offset, size_t dim1_offset,
          int num_yz, struct CompressionMemoryBlocks* memory, size_t* block_pos,
          unsigned char* encoding_buffer)
{
#pragma omp parallel for
  for (int t = 0; t < thread_num; t++) {
    int i = t / (num_yz);
    int j = (t % num_yz) / num_z;
    int k = t % num_z;
    unsigned char* encoding_buffer_pos =
      encoding_buffer + t * max_num_block_elements * sizeof(int);
    size_t enCodeSize = 0;
    size_t offset_x = get_offset(x, i);
    size_t offset_y = get_offset(y, j);
    size_t offset_z = get_offset(z, k);

    size_t current_blockcount_x = get_current_blockcount(x, i);
    size_t current_blockcount_y = get_current_blockcount(y, j);
    size_t current_blockcount_z = get_current_blockcount(z, k);

    size_t current_block_elements =
      current_blockcount_x * current_blockcount_y * current_blockcount_z;
    size_t type_offset = offset_x * dim0_offset +
                         offset_y * current_blockcount_x * dim1_offset +
                         offset_z * current_blockcount_x * current_blockcount_y;
    int* type = (*memory).result_type + type_offset;
    encode(type, current_block_elements, encoding_buffer_pos, &enCodeSize);
    block_pos[t] = enCodeSize;
  }
}

inline unsigned int
quantization_intervals_1D(float* oriData, size_t r1, double realPrecision)
{
  unsigned int quantization_intervals;
  if (optQuantMode == 1) {
    quantization_intervals =
      optimize_intervals_float_1D(oriData, r1, realPrecision);
    updateQuantizationInfo(quantization_intervals);
  } else {
    quantization_intervals = intvCapacity;
  }
  return quantization_intervals;
}
inline unsigned int
quantization_intervals_2D(float* oriData, size_t r1, size_t r2,
                          double realPrecision)
{
  unsigned int quantization_intervals;
  if (optQuantMode == 1) {
    quantization_intervals =
      optimize_intervals_float_2D_opt(oriData, r1, r2, realPrecision);
    printf("2D number of bins: %d\nerror bound %.20f\n", quantization_intervals,
           realPrecision);
    updateQuantizationInfo(quantization_intervals);
  } else {
    quantization_intervals = intvCapacity;
  }
  return quantization_intervals;
}
inline unsigned int
quantization_intervals_3D(float* oriData, size_t r1, size_t r2, size_t r3,
                          double realPrecision)
{
  unsigned int quantization_intervals;
  if (optQuantMode == 1) {
    // quantization_intervals = optimize_intervals_float_3D(oriData, r1,
    // realPrecision);
    quantization_intervals =
      optimize_intervals_float_3D_opt(oriData, r1, r2, r3, realPrecision);
    printf("3D number of bins: %d\nerror bound %.20f\n", quantization_intervals,
           realPrecision);
    // exit(0);
    updateQuantizationInfo(quantization_intervals);
  } else {
    quantization_intervals = intvCapacity;
  }
  return quantization_intervals;
}

inline unsigned char*
write_parallel_compresion_metadata(unsigned char* result_pos, int thread_num,
                                   double realPrecision,
                                   unsigned int quantization_intervals,
                                   struct CompressionMemoryBlocks* memory,
                                   size_t num_blocks, size_t nodeCount,
                                   const unsigned char* treeBytes,
                                   unsigned int treeByteSize)
{
  result_pos = writeIntBigEndian(result_pos, thread_num);
  result_pos = writeDoubleBigEndian(result_pos, realPrecision);
  result_pos = writeIntBigEndian(result_pos, quantization_intervals);
  result_pos = writeIntBigEndian(result_pos, treeByteSize);
  result_pos = writeIntBigEndian(result_pos, nodeCount);
  result_pos = writeBytes(result_pos, (unsigned char*)treeBytes, treeByteSize);
  result_pos =
    writeBytes(result_pos, (unsigned char*)(*memory).unpredictable_count,
               num_blocks * sizeof(unsigned int));
  result_pos = writeBytes(result_pos, (unsigned char*)(*memory).mean,
                          num_blocks * sizeof(float));
  return result_pos;
}

inline unsigned char*
writeBytes(unsigned char* output, const unsigned char* bytes, unsigned int size)
{
  memcpy(output, bytes, size);
  output += size;
  return output;
}
inline unsigned char*
writeDoubleBigEndian(unsigned char* output, double d)
{
  doubleToBytes(output, d);
  output += 8;
  return output;
}
inline unsigned char*
writeIntBigEndian(unsigned char* output, int i)
{
  intToBytes_bigEndian(output, i);
  output += 4;
  return output;
}

inline int
readIntBigEndian(unsigned char** data)
{
  int ret = bytesToInt_bigEndian(*data);
  (*data) += 4;
  return ret;
}

void
Huffman_init_cuda(int* s, size_t length, int thread_num)
{

  size_t i;
  size_t* freq = (size_t*)malloc(thread_num * allNodes * sizeof(size_t));
  memset(freq, 0, thread_num * allNodes * sizeof(size_t));
  size_t block_size = (length - 1) / thread_num + 1;
  size_t block_residue = length - (thread_num - 1) * block_size;

#pragma omp parallel for
  for (int t = 0; t < thread_num; t++) {
    int* s_pos = s + t * block_size;
    size_t* freq_pos = freq + t * allNodes;
    if (t < thread_num - 1) {
      for (size_t i = 0; i < block_size; i++) {
        freq_pos[s_pos[i]]++;
      }
    } else {
      for (size_t i = 0; i < block_residue; i++) {
        freq_pos[s_pos[i]]++;
      }
    }
  }

  size_t* freq_pos = freq + allNodes;
  for (int t = 1; t < thread_num; t++) {
    for (i = 0; i < allNodes; i++) {
      freq[i] += freq_pos[i];
    }
    freq_pos += allNodes;
  }

  for (i = 0; i < allNodes; i++)
    if (freq[i])
      qinsert(new_node(freq[i], i, 0, 0));

  while (qend > 2)
    qinsert(new_node(0, 0, qremove(), qremove()));

  build_code(qq[1], 0, 0, 0);
  free(freq);
}
